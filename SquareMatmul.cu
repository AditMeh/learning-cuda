
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <thread>
#include <chrono>

#define BLOCK_SIZE 4
#define GRID_SIZE 1

__global__ void matmulkernel(float *d_A, float *d_B, float *result, int WIDTH)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < WIDTH) && (col < WIDTH))
    {
        float ret = 0;
        for (int i = 0; i < WIDTH; i++)
        {
            ret += d_A[row * WIDTH + i] * d_B[i * WIDTH + col];
        }
        result[row * WIDTH + col] = ret;
        // printf("(%i, %i) %f\n", row, col, ret);
    }
}

void printvec(float *arr, int len)
{
    for (int j = 0; j < len; j++)
    {
        printf("[");
        for (int i = 0; i < len; i++)
        {
            printf("%f ", arr[j * len + i]);
        }
        printf("]");

        printf("\n");
    }
    printf("\n");
}

void setrand(float *arr, int len)
{
    for (int i = 0; i < len; i++)
    {
        for (int j = 0; j < len; j++)
        {
            arr[i * len + j] = float(rand() % 9);
        }
    }
}

int main(int argc, const char *argv[])
{
    float h_A[(BLOCK_SIZE * 1) * (BLOCK_SIZE * 1)];       // Row major
    float h_B[(BLOCK_SIZE * 1) * (BLOCK_SIZE * 1)];       // Row major
    float h_C[(BLOCK_SIZE * 1) * (BLOCK_SIZE * 1)] = {0}; // Row major

    // Make the first and second vector and put it on the GPU
    setrand(h_A, BLOCK_SIZE * 1);
    setrand(h_B, BLOCK_SIZE * 1);

    int PRINT_FLAG = (int)(*argv[1] - '0');

    if (PRINT_FLAG)
    {
        printvec(h_A, BLOCK_SIZE * 1);
        printvec(h_B, BLOCK_SIZE * 1);
    }
    float *d_A, *d_B, *d_C;

    hipMalloc((void **)&d_A, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float));
    hipMalloc((void **)&d_B, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float));
    hipMalloc((void **)&d_C, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float));

    hipMemcpy(d_A, h_A, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock((BLOCK_SIZE * 1), (BLOCK_SIZE * 1));
    dim3 dimGrid(1, 1);

    matmulkernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, (BLOCK_SIZE * 1));
    hipMemcpy(h_C, d_C, (BLOCK_SIZE * 1) * (BLOCK_SIZE * 1) * sizeof(float), hipMemcpyDeviceToHost);
        

    if (PRINT_FLAG)
    {
        printvec(h_C, BLOCK_SIZE * 1);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}