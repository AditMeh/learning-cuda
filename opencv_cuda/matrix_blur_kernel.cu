
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define BLOCK_SIZE 16

__global__ void matrix_blur_kernel(unsigned char *img, unsigned char *out, int h, int w, int window_size)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int channel = blockIdx.z * blockDim.z + threadIdx.z;
    
    // (i * frame.cols + j) * frame.channels() + k
    if (row < h && col < w)
    {
        int accum = 0;
        int pixels = 0;
        for (int i = -window_size; i <= window_size; i++)
        {
            for (int j = -window_size; j <= window_size; j++)
            {
                int curr_row = row + i;
                int curr_col = col + j;

                if (curr_row >= 0 && curr_row < h && curr_col >= 0 && curr_col < w && channel < 3)
                {
                    accum += img[(curr_row * w + curr_col) * 3 + channel];
                    pixels += 1;
                }
            }
        }

        out[(row * w + col) * 3 + channel] = (unsigned char)(accum / pixels);
    }
}

float invoke_kernel(unsigned char *img, int h, int w, int window_size, int size)
{
    
    unsigned char *d_orig; 
    unsigned char *d_blur;

    hipEvent_t start;
    hipEvent_t stop;
    float ms = 0;

    hipMalloc((void **)&d_orig, size * sizeof(u_char));
    hipMalloc((void **)&d_blur, size * sizeof(u_char));

    hipMemcpy(d_orig, img, size * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 3);
    dim3 dimGrid((h + BLOCK_SIZE - 1) / BLOCK_SIZE, (w + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);


    hipDeviceSynchronize();
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    matrix_blur_kernel<<<dimGrid, dimBlock>>>(d_orig, d_blur, h, w, window_size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    hipMemcpy(img, d_blur, size * sizeof(u_char), hipMemcpyDeviceToHost);

    hipFree(d_orig);
    hipFree(d_blur);
    return ms;
}

